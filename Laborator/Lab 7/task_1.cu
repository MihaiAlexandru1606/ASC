#include "hip/hip_runtime.h"
// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {
	unsigned int threadIdx.x + blockDim.x * blockIdx.x;
	if (i < N)
		if (i % 2 == 0){
			a[i] = 0;
		}else {
			a[i] = 1;
		}
}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {
	unsigned int threadIdx.x + blockDim.x * blockIdx.x;
	if (i  < N)
		a[i] = blockIdx.x;
}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {
	unsigned int threadIdx.x + blockDim.x * blockIdx.x;
        if (i  < N)
                a[i] = threadIdx.x;

}

int main(void) {
	const int num_elements = 1 << 16;
  	const int num_bytes = num_elements * sizeof(int);
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    // ~TODO 1~
    // For each device, show some details in the format below, 
    // then set as active device the first one (assuming there
    // is at least CUDA-capable device). Pay attention to the
    // type of the fields in the hipDeviceProp_t structure.
    //
    // Device number: <i>
    //      Device name: <name>
    //      Total memory: <mem>
    //      Memory Clock Rate (KHz): <mcr>
    //      Memory Bus Width (bits): <mbw>
    // 
    // Hint: look for hipGetDeviceProperties and hipSetDevice in
    // the Cuda Toolkit Documentation. 
    for (int i = 0; i < nDevices; ++i) {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, i);
	printf("Device Number: %d\n", i);
	printf("\tDevice name: %s\n", prop.name);
	printf("\tMemory Clock Rate (KHz): %d\n", prop.memoryClockRate);
	printf("\tMemory Bus Width (bits): %d\n", prop.memoryBusWidth);
    }

    // ~TODO 2~
    // With information from example_2.cu, allocate an array with
    // integers (where a[i] = i). Then, modify the three kernels
    // above and execute them using 4 blocks, each with 4 threads.
    // Hint: num_elements = block_size * block_no (see example_2)
    //
    // You can use the fill_array_int(int *a, int n) function (from utils)
    // to fill your array as many times you want.
	int *host_array = (int*) malloc(num_bytes);
	int *device_array;
	hipMalloc( (void **) &a_device, num_bytes);
	
	const size_t block_size = 4;
	size_t blocks_no = num_elements / block_size;
	
	if (num_elements % block_size) 
		++blocks_no;

	

    // ~TODO 3~
    // Execute kernel_parity_id kernel and then copy from 
    // the device to the host; call hipDeviceSynchronize()
    // after a kernel execution for safety purposes.
    //
    // Uncomment the line below to check your results
	kernel_parity_id<<blocks_no, block_size>> (host_array, num_elements);
	hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
	check_task_1(3, host_array);
 
	

    // ~TODO 4~
    // Execute kernel_block_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results
	kernel_block_id<<<blocks_no, block_size>>>(device_array, num_elements);
	hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
    	check_task_1(4, host_array);

    // ~TODO 5~
    // Execute kernel_thread_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results
	kernel_thread_id<<<blocks_no, block_size>>>(device_array, num_elements);
        hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

    	check_task_1(5, host_array);
	

    // TODO 6: Free the memory
    	free(host_array);
	hipFree(device_array);
    return 0;
}

