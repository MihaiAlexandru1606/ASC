#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  	// Avoid accessing out of bounds elements
  	if (i < N) {
    		c[i] = a[i] + b[i];
  	}    
}

int main(void) {
    	hipSetDevice(0);
    	int N = 1 << 20;
	const int num_elements = N;
  	const int num_bytes = num_elements * sizeof(float);
    	float *host_array_a = 0;
    	float *host_array_b = 0;
    	float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays
		

    // TODO 2: Allocate the device's arrays

    // TODO 3: Check for allocation errors

	// Allocating the host array
  	host_array_a = (float *) malloc(num_bytes);
	host_array_b = (float *) malloc(num_bytes);
	host_array_c = (float *) malloc(num_bytes);

	
  	// Allocating the device's array; notice that we use a special
  	// function named hipMalloc that takes the reference of the
  	// pointer declared above and the number of bytes.
  	hipMalloc((void **) &device_array_a, num_bytes);
	hipMalloc((void **) &device_array_b, num_bytes);
	hipMalloc((void **) &device_array_c, num_bytes);

  	// If any memory allocation failed, report an error message
  	if (host_array_a == 0 || host_array_b == 0 || host_array_c == 0 || device_array_a == 0 || device_array_b == 0 || device_array_c == 0) {
    		printf("[HOST] Couldn't allocate memory\n");
    		return 1;
  	}

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.
	fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);


    // TODO 5: Copy the host's arrays to device
	hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).
	const size_t block_size = 256;
  	size_t blocks_no = num_elements / block_size;

	if (num_elements % block_size) 
		++blocks_no;

	add_arrays<<<blocks_no, block_size>>>(device_array_a, device_array_b, device_array_c, N);
	
    // TODO 7: Copy back the results and then uncomment the checking function
	hipDeviceSynchronize();	
hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost);

    check_task_2(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory
   	free(host_array_a);
	free(host_array_b);
	free(host_array_c);
	hipFree(device_array_a);
	hipFree(device_array_b);
	hipFree(device_array_c);

    return 0;
}

